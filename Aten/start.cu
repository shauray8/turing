#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>

__global__ void add_int(int* a, int* b, int count){
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id < count){
    a[id] *= b[id];
  }
}

void normal_stuff(int* a, int* b, int count){
  for(int i=0; i<count; i++){
    a[i] *= b[i];
  }
}

int main(){
  srand(time(NULL));
  int count = 4096;
  int *h_a = new int[count];
  int *h_b = new int[count];

  for(int i=0; i<count; i++){
    h_a[i] = rand()%1000;
    h_b[i] = rand()%1000;
  }
  
  int *d_a, *d_b;

  if(hipMalloc(&d_a, sizeof(int)*count) != hipSuccess){
    printf("take me to the moon BITCH ! A");
    return 0;
  }
  if(hipMalloc(&d_b, sizeof(int)*count) != hipSuccess){
    printf("take me to the moon BITCH ! B");
    hipFree(d_a);
    return 0;
  }

  if(hipMemcpy(d_a, h_a, sizeof(int)*count, hipMemcpyHostToDevice) != hipSuccess){
    printf("fly me to the sky !! ");
    hipFree(d_a);
    hipFree(d_b);
    return 0;
  }
  if(hipMemcpy(d_b, h_b, sizeof(int)*count, hipMemcpyHostToDevice) != hipSuccess){
    printf("fly me to another sky !! ");
    hipFree(d_a);
    hipFree(d_b);
    return 0;
  }

  add_int<<<count / 256 + 1, 256>>>(d_a, d_b, count);
  //normal_stuff(h_a, h_b, count);
  
  if(hipMemcpy(h_a, d_a, sizeof(int)*count, hipMemcpyDeviceToHost) != hipSuccess){
    printf("make me a pancake !! ");
    delete[] h_a;
    delete[] h_b;
    hipFree(d_a);
    hipFree(d_b);
    return 0;
  }

  for(int j=0;j<5;j++){
    printf("the answer is : %d %d\n",h_a[j], h_b[j]);
  }

  hipFree(d_a);
  hipFree(d_b);

  delete[] h_a;
  delete[] h_b;

  hipDeviceReset();

  return 0;
  
}
